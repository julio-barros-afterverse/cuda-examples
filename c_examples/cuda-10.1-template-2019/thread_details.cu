
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//
//__global__ void hello_cuda() {
//	printf("Hello CUDA world \n");
//}
//
//__global__ void print_threadIds() {
//	printf("threadIdx.x: %d | threadIdx.y: %d | threadIdx.z: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
//
//	//printf("blockIdx.x: %d | blockIdx.y: %d | blockIdx.z: %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
//
//	//printf("blockDim.x: %d | blockDim.y: %d | blockDim.z: %d\n", blockDim.x, blockDim.y, blockDim.z);
//
//	//printf("gridDim.x: %d | gridDim.y: %d | gridDim.z: %d\n", gridDim.x, gridDim.y, gridDim.z);
//}
//
//int main() {
//	dim3 block(32, 32);
//	dim3 grid(4, 4);
//
//	print_threadIds <<<grid, block>>>(); 
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}